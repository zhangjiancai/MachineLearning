
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
void CuDNNLCNLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();

#ifdef USE_CNMEM
  MemoryHandler::mallocGPU(&this->tempData1, this->tempDataSize);
  MemoryHandler::mallocGPU(&this->tempData2, this->tempDataSize);
#endif

  CUDNN_CHECK(cudnnDivisiveNormalizationForward(
        Caffe::cudnn_handle(), norm_desc_, CUDNN_DIVNORM_PRECOMPUTED_MEANS,
        cudnn::dataType<Dtype>::one,
        bottom_desc_, bottom_data,
        NULL,  // srcMeansData
        this->tempData1, this->tempData2,
        cudnn::dataType<Dtype>::zero,
        top_desc_, top_data) );

#ifdef USE_CNMEM
  MemoryHandler::freeGPU(this->tempData1);
  MemoryHandler::freeGPU(this->tempData2);
  this->tempData1 = NULL;
  this->tempData2 = NULL;
#endif
}

template <typename Dtype>
void CuDNNLCNLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* top_data = top[0]->gpu_data();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();

#ifdef USE_CNMEM
  MemoryHandler::mallocGPU(&this->tempData1, this->tempDataSize);
  MemoryHandler::mallocGPU(&this->tempData2, this->tempDataSize);
#endif

  CUDNN_CHECK(cudnnDivisiveNormalizationBackward(
        Caffe::cudnn_handle(), norm_desc_, CUDNN_DIVNORM_PRECOMPUTED_MEANS,
        cudnn::dataType<Dtype>::one,
        bottom_desc_, bottom_data,
        NULL, top_diff,  // NULL - srcMeansData
        this->tempData1, this->tempData2,
        cudnn::dataType<Dtype>::zero,
        bottom_desc_, bottom_diff,
        NULL) );

#ifdef USE_CNMEM
  MemoryHandler::freeGPU(this->tempData1);
  MemoryHandler::freeGPU(this->tempData2);
  this->tempData1 = NULL;
  this->tempData2 = NULL;
#endif
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNLCNLayer);

}  // namespace caffe
#endif
