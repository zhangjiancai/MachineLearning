
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/gpu_memory.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

// Those defines serve single purpose to keep sane C++ formatting
// in presence of <80 characters rule
#define cudnnConvFwd                       cudnnConvolutionForward
#define cudnnConvBwdBias                   cudnnConvolutionBackwardBias
#define cudnnConvBwdFilter                 cudnnConvolutionBackwardFilter
#define cudnnConvBwdData                   cudnnConvolutionBackwardData

namespace caffe {

  __global__ void sync_conv_groups() { }

  template <typename Dtype>
  void CuDNNConvolutionLayer<Dtype>::
  Forward_gpu(const vector<Blob<Dtype>*>& bottom,
              const vector<Blob<Dtype>*>& top) {
    const Dtype* weight = this->blobs_[0]->gpu_data();
    for (int i = 0; i < bottom.size(); ++i) {
      const Dtype* bottom_data = bottom[i]->gpu_data();
      Dtype* top_data = top[i]->mutable_gpu_data();

      // Test free space and force reshape if allocations have changed
      size_t workspace_limit_bytes, total_memory;
      gpu_memory::getInfo(&workspace_limit_bytes, &total_memory);
      if (workspace_fwd_sizes_[i] > workspace_limit_bytes) {
          this->Reshape(bottom, top);
      }

      // !!!! Not safe if group_ > 1 !!!!
      workspace.reserve(workspace_fwd_sizes_[i]);

      // Forward through cuDNN in parallel over groups.
      for (int g = 0; g < this->group_; g++) {
          // Filters.
        CUDNN_CHECK(cudnnConvFwd(Caffe::cudnn_handle(),
                                 cudnn::dataType<Dtype>::one,
                                 bottom_descs_[i],
                                 bottom_data + bottom_offset_ * g,
                                 filter_desc_,
                                 weight + this->weight_offset_ * g,
                                 conv_descs_[i],
                                 fwd_algo_[i],
                                 workspace.data(),
                                 workspace.size(),
                                 cudnn::dataType<Dtype>::zero,
                                 top_descs_[i],
                                 top_data + top_offset_ * g));

        // Bias.
        if (this->bias_term_) {
          const Dtype* bias_data = this->blobs_[1]->gpu_data();
          CUDNN_CHECK(cudnnAddTensor(Caffe::cudnn_handle(),
                                     cudnn::dataType<Dtype>::one,
                                     bias_desc_,
                                     bias_data + bias_offset_ * g,
                                     cudnn::dataType<Dtype>::one,
                                     top_descs_[i],
                                     top_data + top_offset_ * g));
        }
      }

      workspace.release();
      // Synchronize the work across groups, each of which went into its own
      // stream, by launching an empty kernel into the default (null) stream.
      // NOLINT_NEXT_LINE(whitespace/operators)
      CUDA_CHECK(cudaStreamSynchronize(cudaStreamLegacy));
    }
  }

  template <typename Dtype>
  void
  CuDNNConvolutionLayer<Dtype>::
  Backward_gpu(const vector<Blob<Dtype>*>& top,
               const vector<bool>& propagate_down,
               const vector<Blob<Dtype>*>& bottom) {
    const Dtype* weight = NULL;
    Dtype* weight_diff = NULL;


    if (this->param_propagate_down_[0]) {
      weight = this->blobs_[0]->gpu_data();
      weight_diff = this->blobs_[0]->mutable_gpu_diff();
    }
    Dtype* bias_diff = NULL;

    if (this->bias_term_ && this->param_propagate_down_[1]) {
      bias_diff = this->blobs_[1]->mutable_gpu_diff();
    }

    for (int i = 0; i < top.size(); ++i) {
        const Dtype* top_diff = top[i]->gpu_diff();

        // Test free space and force reshape if allocations have changed
        size_t workspace_limit_bytes, total_memory;
        gpu_memory::getInfo(&workspace_limit_bytes, &total_memory);
        if (workspace_bwd_filter_sizes_[i] > workspace_limit_bytes ||
           workspace_bwd_data_sizes_[i] > workspace_limit_bytes) {
            this->Reshape(bottom, top);
        }

        // To remove pressure on allocator, allocate the larger of the
        // workspaces needed for the following steps
        size_t workspace_reserve = workspace_bwd_filter_sizes_[i] >
            workspace_bwd_data_sizes_[i] ?
            workspace_bwd_filter_sizes_[i] : workspace_bwd_data_sizes_[i];

        // !!!! Not safe if group_ > 1 !!!!
        workspace.reserve(workspace_reserve);

        // Backward through cuDNN in parallel over groups and gradients.
        for (int g = 0; g < this->group_; g++) {
            // Gradient w.r.t. bias.
            if (this->bias_term_ && this->param_propagate_down_[1]) {
                CUDNN_CHECK(cudnnConvBwdBias(Caffe::cudnn_handle(),
                                             cudnn::dataType<Dtype>::one,
                                             top_descs_[i],
                                             top_diff + top_offset_ * g,
                                             cudnn::dataType<Dtype>::one,
                                             bias_desc_,
                                             bias_diff + bias_offset_ * g));
            }

            // Gradient w.r.t. weights.
            if (this->param_propagate_down_[0]) {
                const Dtype* bottom_data = bottom[i]->gpu_data();
                CUDNN_CHECK(cudnnConvBwdFilter(Caffe::cudnn_handle(),
                                          cudnn::dataType<Dtype>::one,
                                          bottom_descs_[i],
                                          bottom_data + bottom_offset_ * g,
                                          top_descs_[i],
                                          top_diff + top_offset_ * g,
                                          conv_descs_[i],
                                          bwd_filter_algo_[i],
                                          workspace.data(),
                                          workspace.size(),
                                          cudnn::dataType<Dtype>::one,
                                          filter_desc_,
                                          weight_diff + weight_offset_ * g));
            }

            // Gradient w.r.t. bottom data.
            if (propagate_down[i]) {
                if (weight == NULL) {
                    weight = this->blobs_[0]->gpu_data();
                }
                Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
                CUDNN_CHECK(cudnnConvBwdData(Caffe::cudnn_handle(),
                                             cudnn::dataType<Dtype>::one,
                                             filter_desc_,
                                             weight + this->weight_offset_ * g,
                                             top_descs_[i],
                                             top_diff + top_offset_ * g,
                                             conv_descs_[i],
                                             bwd_data_algo_[i],
                                             workspace.data(),
                                             workspace.size(),
                                             cudnn::dataType<Dtype>::zero,
                                             bottom_descs_[i],
                                             bottom_diff + bottom_offset_ * g));
            }
        }

        workspace.release();
        // Synchronize the work across groups, each of which went into its own
        // stream, by launching an empty kernel into the default (null) stream.
        // NOLINT_NEXT_LINE(whitespace/operators)
        CUDA_CHECK(cudaStreamSynchronize(cudaStreamLegacy));
    }
  }

  INSTANTIATE_LAYER_GPU_FUNCS(CuDNNConvolutionLayer);

}  // namespace caffe
#endif
