#include "hip/hip_runtime.h"
#include "nvmatrix/include/nvmatrix_kernels.cuh"
#include "nvmatrix/include/nvmatrix_operators.cuh"
#include <THC.h>
#include <algorithm>
#include "hip/hip_runtime_api.h"

extern "C" {

  void addBias(THCState* state, THCudaTensor* output, THCudaTensor* bias) {
    int width = output->size[1];
    int height = output->size[0];
    float *odata = THCudaTensor_data(state, output);
    float *bdata = THCudaTensor_data(state, bias);
    dim3 threads(ADD_VEC_THREADS_X, ADD_VEC_THREADS_Y);
    dim3 blocks(std::min(512, DIVUP(width, ADD_VEC_THREADS_X)),
                std::min(NUM_BLOCKS_MAX, DIVUP(height, ADD_VEC_THREADS_Y)));
    hipStream_t stream = THCState_getCurrentStream(state);
    kColVectorOp<NVMatrixBinaryOps::Add><<<blocks, threads, 0, stream>>>(
      odata, bdata, odata, width, height,
      output->stride[0], output->stride[0],
      NVMatrixBinaryOps::Add());
    getLastCudaError("Kernel execution failed");
  }

  void gradBias(THCState* state, THCudaTensor* gradOutput, THCudaTensor* gradBias, float scale) {
    dim3 threads(AWR_NUM_THREADS);
    dim3 blocks(1, gradOutput->size[0]);
    hipStream_t stream = THCState_getCurrentStream(state);
    kAggRows_wholerow_nosync<<<blocks, threads, 0, stream>>>(
      THCudaTensor_data(state, gradOutput),
      THCudaTensor_data(state, gradBias),
      gradOutput->size[1],
      gradOutput->size[0],
      NVMatrixAggs::Sum(),
      NVMatrixOps::Identity(),
      NVMatrixBinaryOps::SecondScaled(scale));
  }

  // output = weights, input = wtemp
  void addSumCols(THCState* state, THCudaTensor*output, THCudaTensor*input) {
    int width = input->size[1];
    int height = input->size[0];
    THCudaTensor_resize2d(state, output, 1, width);
    hipStream_t stream = THCState_getCurrentStream(state);
    if ((height <= 2048 || width >= 4096)) {
      int numBlocks = DIVUP(width, NUM_SUM_COLS_THREADS_PER_BLOCK);
      THAssert(numBlocks * NUM_SUM_COLS_THREADS_PER_BLOCK >= width);
      THAssert(numBlocks < NUM_BLOCKS_MAX);
      hipTextureObject_t texInput = THCudaTensor_getTextureObject(state, input);
      kDumbAggCols<NVMatrixAggs::Sum, NVMatrixOps::Identity, NVMatrixBinaryOps::SecondScaled>
        <<<numBlocks,NUM_SUM_COLS_THREADS_PER_BLOCK, 0, stream>>>(
          texInput,
          THCudaTensor_data(state, output), width, height,
          NVMatrixAggs::Sum(), NVMatrixOps::Identity(),
          NVMatrixBinaryOps::SecondScaled(1.0));
      getLastCudaError("kDumbAggCols: Kernel execution failed");
      checkCudaErrors(hipDestroyTextureObject(texInput));
    } else { // Specialize the case when we have very long columns and few of them
      const int sumLength = 128;
      THCudaTensor* tmp = THCudaTensor_newWithSize2d(state, DIVUP(height, sumLength), width);
      int numBlocksX = DIVUP(width, NUM_SUM_COLS_THREADS_PER_BLOCK);
      int numBlocksY = DIVUP(height, sumLength);
      dim3 blocks(numBlocksX, numBlocksY);
      dim3 threads(NUM_SUM_COLS_THREADS_PER_BLOCK);
      hipTextureObject_t texInput = THCudaTensor_getTextureObject(state, input);
      kAggCols<NVMatrixAggs::Sum, NVMatrixOps::Identity><<<blocks,threads, 0, stream>>>(
        texInput, THCudaTensor_data(state, tmp),
        width, height, sumLength, NVMatrixAggs::Sum(), NVMatrixOps::Identity());
      getLastCudaError("kAggCols: Kernel execution failed");
      checkCudaErrors(hipDestroyTextureObject(texInput));

      int numBlocks = DIVUP(width, NUM_SUM_COLS_THREADS_PER_BLOCK);
      hipTextureObject_t texTmp = THCudaTensor_getTextureObject(state, tmp);
      kDumbAggCols<NVMatrixAggs::Sum, NVMatrixOps::Identity, NVMatrixBinaryOps::SecondScaled>
        <<<numBlocks,NUM_SUM_COLS_THREADS_PER_BLOCK, 0, stream>>>(
          texTmp, THCudaTensor_data(state, output), width, height,
          NVMatrixAggs::Sum(), NVMatrixOps::Identity(),
          NVMatrixBinaryOps::SecondScaled(1.0));
      getLastCudaError("kDumbAggCols: Kernel execution failed");
      checkCudaErrors(hipDestroyTextureObject(texTmp));
      THCudaTensor_free(state, tmp);
    }
  }
}
